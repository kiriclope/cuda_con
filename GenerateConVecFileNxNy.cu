#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "mycurand.h"
#include "librairies.h"
#include "cuPrintf.cu"
#include "devFunctionProtos.h"
#include "devHostConstants.h"
#include "CudaFunc.cu"
#include "Matrix_Utils.cu"
#include "GenConProbDistDepMat.cu"

///////////////////////////////////////////////////////////////////  

int main(int argc, char *argv[]) {
  
  char* AtoB ;
  AtoB =  (char *) malloc( strlen("EE") ) ;

  if(IF_LARGE)
    AtoB = argv[1] ;
  
  unsigned long int *nbN, *Cpt ;
  nbNeurons(nbN) ;
  CptNeurons(nbN, Cpt) ;
  
  // ///////////////////////////////////////////////////////////////////    
  
  unsigned long nChunks = 1, deviceId = 0 ;
  unsigned long maxNeurons = NX_NEURONS ; 

  ///////////////////////////////////////////////////////////////////
  
  hipDeviceProp_t prop;
  unsigned long long maxMem = 12079136768 ; 

  cudaCheck(hipGetDeviceProperties(&prop, deviceId)) ;
  printf("Global Mem = %ld, ", prop.totalGlobalMem) ; 
  maxMem = prop.totalGlobalMem;

  if( maxMem < (unsigned long long) (NX_NEURONS * NY_NEURONS * 4 + NX_NEURONS * 4) ) { 
    while( maxMem < (unsigned long) ( (NX_NEURONS / nChunks) * NY_NEURONS * 4   + NX_NEURONS * 5 ) ) 
      nChunks += 1 ;     
    if( nChunks % 2 !=0 ) 
      nChunks += 1 ; 
  }

  maxNeurons = (unsigned long) NX_NEURONS / nChunks ; //divide the presynaptic into nChunks 

  if(IF_CHUNKS) { 
    nChunks = NCHUNKS ; 
    maxNeurons = MAXNEURONS ; 
  }

  printf(" maxNeurons = %lu, nChunks = %lu\n", maxNeurons, nChunks); 

  /////////////////////////////////////////////////////////////////// 

  /* choose 256 threads per block for high occupancy */
  int ThreadsPerBlock = N_THREADS ; 
  int BlocksPerGrid = ( NX_NEURONS + ThreadsPerBlock-1 ) / ThreadsPerBlock ; 
  
  if(BlocksPerGrid > 65536) { 
    printf("BlocksPerGrid exceds valid number of allowed blocks of 65536"); 
    exit(-1); 
  } 
  
  hiprandState *devStates; 
  cudaCheck(hipMalloc((void **)&devStates, NX_NEURONS * sizeof(hiprandState))); 
  
  printf("Setup kernel ... \n"); 
  setup_kernel<<<BlocksPerGrid, ThreadsPerBlock>>>(devStates, time(NULL));
  cudaCheckLastError("setup_kernel failed\n"); 

  /////////////////////////////////////////////////////////////////// 

  unsigned long long chunckSize = (unsigned long long) ( NX_NEURONS / nChunks * NY_NEURONS ) ; 
  printf("chunckSize = %llu, ", chunckSize);

  BlocksPerGrid = (maxNeurons + ThreadsPerBlock - 1) / ThreadsPerBlock ; 
  printf("Threads per block : %d, Blocks per grid : %d \n", ThreadsPerBlock, BlocksPerGrid);

  // ///////////////////////////////////////////////////////////////////    

  float *dev_conVecPtr, *dev_preFactor ; //*preFactor = NULL ; 
  float *fullConVec = NULL, *conVec = NULL ; 
  unsigned long *IdPost ; 
  int *nbPost ; 

  ///////////////////////////////////////////////////////////////////
  
  fullConVec = (float *) malloc((unsigned long long) NX_NEURONS * NY_NEURONS * sizeof(float)) ;

  //NX_NEURONS presynaptic and NY_NEURONS postsynaptic 
  IdPost = (unsigned long *) malloc( (unsigned long long) NX_NEURONS * ( K + sqrt(K) ) * sizeof( unsigned long ) ) ; // idx of the postsynaptic neurons from 1 to NY_NEURONS on average K * NX_NEURONS +- sqrt(K) 
  nbPost = (int *) malloc( (unsigned long long) NX_NEURONS * sizeof(int)) ; // number of postsynaptic neurons of the NX_NEURONS

  int **nbPreSab = (int **)malloc(nbpop * sizeof(int *) ) ; 
  for(int i=0; i<nbpop; i++) 
    nbPreSab[i] = (int *) malloc(nbpop * sizeof(int) ) ; 
  for(int i=0; i<nbpop; i++) 
    for(int j=0;j<nbpop;j++) 
      nbPreSab[i][j] = 0 ;
  
  ////////////////////////////////////////////////////////////////////    

  cudaCheck(hipHostMalloc((void **)&conVec, (unsigned long long) chunckSize * sizeof(float))) ;  
  cudaCheck(hipMalloc((void **)&dev_conVecPtr, (unsigned long long) chunckSize * sizeof(float))) ; 
  cudaCheck(hipMalloc((void **)&dev_preFactor, (unsigned long long) nbpop * NX_NEURONS * sizeof(float))) ; 
  
  // cudaCheck(hipHostMalloc((void **)&preFactor, 2 * N_NEURONS * sizeof(float))); 

  ///////////////////////////////////////////////////////////////////

  enum ConMat_type {
    random, distDependent
  };

  ConMat_type conMatType = random ; 
  if(IF_SPACE) {
    printf("Generating Spatial Matrix ... \n") ; 
    conMatType = distDependent ;
  }
  else 
    if(IF_RING) 
      printf("Generating Ring ... \n") ; 
    else 
      printf("Generating Random Matrix ... \n") ; 
  
  if(IF_SPEC) 
    printf("with specific connections ... \n") ; 

  if(IF_AUTA) 
    printf("with autapses ... \n") ; 
  
  ///////////////////////////////////////////////////////////////////
  
  double *host_Sigma ;
  cudaCheck(hipHostMalloc((void **)&host_Sigma,  nbpop * sizeof(double))) ; 
  for(int j=0;j<nbpop;j++) 
    host_Sigma[j] = Sigma[j] ; 
  if(IF_RING & IF_SPEC) 
    for(int j=0;j<nbpop;j++) 
      host_Sigma[j] = host_Sigma[j]/sqrt(K) ; 

  double *host_Dij ; 
  cudaCheck(hipHostMalloc((void **)&host_Dij,  nbpop * sizeof(double))) ; 
  if(IF_RING & IF_SPEC) 
    for(int j=0;j<nbpop*nbpop;j++) 
      host_Dij[j] = Dij[j] ; 
    

  if(IF_SPACE || IF_RING) { 
    printf("Sigma ") ; 
    for(int j=0;j<nbpop;j++)  
      printf("%.4f ",Sigma[j]) ; 
    printf("\n") ; 
  } 

  cudaPrintfInit(); 

  switch(conMatType) {
    
  case random: 
    
    for(unsigned long i = 0; i < nChunks; i++) { 

      initConVec<<<BlocksPerGrid, ThreadsPerBlock>>>(dev_conVecPtr, i, maxNeurons);
      
      printf("Generating chunk %lu ... \n", i) ; fflush(stdout) ;
      
      printf(" Generating Binary Matrix ...\n") ;
      if(IF_RING) {
	KernelGenConRing<<<BlocksPerGrid, ThreadsPerBlock>>>(devStates, dev_conVecPtr, i, maxNeurons, nbN, Cpt, host_Sigma, host_Dij) ; 
	KernelGenDistDepConMat<<<BlocksPerGrid, ThreadsPerBlock>>>(devStates, dev_conVecPtr, i, maxNeurons) ; 
	cudaPrintfDisplay(stdout, true);
      }
      else {
	if(IF_AUTA) 
	  kernelGenConAuta<<<BlocksPerGrid, ThreadsPerBlock>>>(devStates, dev_conVecPtr, i, maxNeurons, nbN) ; 

	kernelGenConMat<<<BlocksPerGrid, ThreadsPerBlock>>>(devStates, dev_conVecPtr, i, maxNeurons, nbN) ; 
	cudaPrintfDisplay(stdout, true);
	
      }
      
      printf("  Copy dev to Host ... \n") ; 
      cudaCheck(hipMemcpy(conVec, dev_conVecPtr, (unsigned long long) chunckSize * sizeof(float), hipMemcpyDeviceToHost));
      
      for(unsigned long j = 0; j < chunckSize ; j++) {
	fullConVec[j + chunckSize * i] = (float) conVec[j] ; 
	// printf("# %llu Con %f fullConVec %f \n", j + chunckSize * i, conVec[j], fullConVec[j + chunckSize * i]) ;
	conVec[j] = 0 ;
      }
    }

    cudaPrintfEnd();

    break;
    
  case distDependent:
    initPreFactor<<<BlocksPerGrid, ThreadsPerBlock>>>(dev_preFactor);
    
    for(unsigned long i = 0; i < nChunks; i++) { 

      initConVec<<<BlocksPerGrid, ThreadsPerBlock>>>(dev_conVecPtr, i, maxNeurons) ;

      printf("Generating chunk %lu ... \n", i); fflush(stdout);
	
      printf(" Generating Probabilty Matrix ...\n");
      if(DIMENSION==1)
	KernelGenConProbMat<<<BlocksPerGrid, ThreadsPerBlock>>>(dev_conVecPtr,i,maxNeurons,nbN,Cpt,host_Sigma) ; 
      else
	KernelGenConProbMat2D<<<BlocksPerGrid, ThreadsPerBlock>>>(dev_conVecPtr,i,maxNeurons,nbN,Cpt,host_Sigma) ; 
      
      printf("  Generating preFactor ...\n");
      KernelConProbPreFactor<<<BlocksPerGrid, ThreadsPerBlock>>>(dev_conVecPtr, dev_preFactor, i, maxNeurons) ;      

      printf("   Copy dev to Host ...\n") ;
      cudaCheck(hipMemcpy(conVec, dev_conVecPtr, (unsigned long long) chunckSize * sizeof(float), hipMemcpyDeviceToHost)) ;

      for(unsigned long j = 0; j < chunckSize ; j++) {
	fullConVec[j + chunckSize * i] = conVec[j] ; 
	
	// if(conVec[j]!=1) {
	//   printf("\n ERRROR Chunk %llu conVec[%llu] = %.3f \n", i, j, conVec[j] ) ;
	//   exit(-1) ;
	// }
	
	conVec[j] = 0 ;
      }      

    }
    
    // printf("Copy preFactor to Host ...") ; 
    // cudaCheck(hipMemcpy(preFactor, dev_preFactor, 2 * N_NEURONS * sizeof(float), hipMemcpyDeviceToHost) ) ; 
    // printf(" Done\n ") ; 
    
    // printf(" Check preFactor ...") ; 
    // for(int j=0;j<2*N_NEURONS;j++)
    //   if(preFactor[j]!=N_NEURONS/nbpop) {
    // 	printf("ERROR clm %d prefactor %.3f \n", j, preFactor[j]) ;
    // 	exit(-1) ;
    //   }
    // printf(" %.0f ", preFactor[0]) ;
    // printf(" Done\n") ; 

    for(unsigned long i = 0; i < nChunks; i++) { 

      printf("Generating chunk %lu ... \n", i); fflush(stdout);
      
      initConVec<<<BlocksPerGrid, ThreadsPerBlock>>>(dev_conVecPtr, i, maxNeurons) ;

      for(unsigned long j = 0; j < chunckSize ; j++) 
	conVec[j] = fullConVec[j + chunckSize * i] ; 

      printf(" Copy Host to dev ...\n") ;
      cudaCheck(hipMemcpy(dev_conVecPtr, conVec, (unsigned long long) chunckSize * sizeof(float), hipMemcpyHostToDevice)) ;
      
      printf("  Generating Normalized Matrix ...\n") ;
      KernelConProbNorm<<<BlocksPerGrid, ThreadsPerBlock>>>(dev_conVecPtr, dev_preFactor, i, maxNeurons) ; 
            
      printf("   Generating Binary Matrix ...\n") ; 
      KernelGenDistDepConMat<<<BlocksPerGrid, ThreadsPerBlock>>>(devStates, dev_conVecPtr, i, maxNeurons) ; 
            
      cudaCheck(hipMemcpy(conVec, dev_conVecPtr, (unsigned long long) chunckSize * sizeof(float), hipMemcpyDeviceToHost)) ; 
      
      for(unsigned long j = 0; j < chunckSize ; j++) { 

	// if(normConVec[j]!=N_NEURONS/nbpop) { 
	//   printf("\n ERRROR Chunk %llu normConVec[%llu] = %.0f \n", i, j, conVec[j] ) ; 
	//   exit(-1) ; 
	// }

	fullConVec[j + chunckSize * i] = conVec[j] ; 
	conVec[j] = 0 ; 
      }
      
    }
    
    break ; 
    
  default: 
    for(unsigned long i = 0; i < nChunks; i++) 
      kernelGenConMat<<<BlocksPerGrid, ThreadsPerBlock>>>(devStates, dev_conVecPtr, i, maxNeurons, nbN);
  }
  
  printf("Free devPtr ... "); 

  hipFree(dev_conVecPtr); 
  hipFree(dev_preFactor); 
  
  hipHostFree(host_Sigma); 
  hipHostFree(conVec); 

  printf("Done\n") ; 
  
  /////////////////////////////////////////////////////////////////// 

  // ///////////////////////////////////////////////////////////////////    
  // // On CPU 
  // ///////////////////////////////////////////////////////////////////    

  ////////////////////////////////////////////////////////////////////    
  
  unsigned long *idxPost = (unsigned long *) malloc( (unsigned long long) NX_NEURONS * sizeof(unsigned long) ) ; // idx of the post neurons 
  idxPost[0] = 0 ; 
  
  char *path = '\0';
  CreatePath(path) ; 

  if(IF_SPARSEVEC) {
    printf("Generating vectors nbPost & IdPost ... "); 
    
    unsigned long counter = 0 ; 

    if(NX_NEURONS==NY_NEURONS) 
      for(int i=0;i<nbpop;i++) 
	for(unsigned long k=Cpt[i];k<Cpt[i+1];k++) { //Presynaptic neurons
	  for(int j=0;j<nbpop;j++) 
	    for(unsigned long l=Cpt[j];l<Cpt[j+1];l++) //Postsynaptic neurons
	      if(fullConVec[k + NY_NEURONS * l]) { // k-->l column to row 
		IdPost[counter] = l ;
		nbPost[k]++ ;
		nbPreSab[j][i]++ ;
		counter+=1 ;
	      }   
	  // printf("PresId %d, nPost %d \r",k,nbPost[k]); 
	}
    else {
      printf("NX %llu NY %llu... ", NX_NEURONS, NY_NEURONS); 
      nbN[0] = NX_NEURONS ;
      for(unsigned long k=0;k<NX_NEURONS;k++) { //Presynaptic neurons
	for(unsigned long l=0;l<NY_NEURONS;l++) //Postsynaptic neurons
	  if(fullConVec[k + NX_NEURONS * l]) { // k-->l column to row 
	    IdPost[counter] = l ;
	    nbPost[k]++ ;
	    nbPreSab[0][0]++ ; 
	    counter+=1 ;
	  }
      }
      // printf("PresId %d, nPost %d \r",k,nbPost[k]);      
    }
      
    // Average number of Presynaptic neurons      
    CheckPres(path,nbN,nbPreSab) ; 
    free(nbPreSab);

    ///////////////////////////////////////////////////////////////////    
    // Writing to File
    ///////////////////////////////////////////////////////////////////

    if(IF_LARGE)
      WritetoFileLarge(path,IdPost,nbPost,idxPost,AtoB) ;
    else
      WritetoFile(path,IdPost,nbPost,idxPost) ;
  }

  free(IdPost);
  free(idxPost);
  free(nbPost);

  ///////////////////////////////////////////////////////////////////    
  // Writing Complete Matrix
  ///////////////////////////////////////////////////////////////////

  if(IF_MATRIX) {
    // CheckSparseVec(path) ;
    WriteMatrix(path,fullConVec) ;
  }

  printf("Free Host ptr ... ") ;

  hipHostFree(nbN); 
  hipHostFree(Cpt); 

  //free(fullConVec);

  printf("Done\n") ;

  return 0 ;
  
}